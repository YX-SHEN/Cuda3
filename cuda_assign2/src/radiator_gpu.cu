#include "hip/hip_runtime.h"
#include "radiator_gpu.h"
#include "cuda_helper.h"
#include <cmath>
#include <cstdio>
#include <algorithm>

// ---------------- Texture-based Propagation Kernel ----------------
__global__ void propagate_kernel(hipTextureObject_t tex_in, float* out, int n, int m) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col >= m) return;

    if (col == 0) {
        out[row * m + col] = tex2D<float>(tex_in, col + 0.5f, row + 0.5f);
        return;
    }

    int jm2 = (col - 2 + m) % m;
    int jm1 = (col - 1 + m) % m;
    int jp1 = (col + 1) % m;
    int jp2 = (col + 2) % m;

    float sum = 1.60f * tex2D<float>(tex_in, jm2 + 0.5f, row + 0.5f)
              + 1.55f * tex2D<float>(tex_in, jm1 + 0.5f, row + 0.5f)
              + 1.00f * tex2D<float>(tex_in, col  + 0.5f, row + 0.5f)
              + 0.60f * tex2D<float>(tex_in, jp1 + 0.5f, row + 0.5f)
              + 0.25f * tex2D<float>(tex_in, jp2 + 0.5f, row + 0.5f);

    out[row * m + col] = sum / 5.0f;
}

__global__ void average_kernel(const float* matrix, float* averages, int n, int m) {
    extern __shared__ float sdata[];
    int row = blockIdx.x;
    int tid = threadIdx.x;
    float local_sum = 0.0f;
    for (int i = tid; i < m; i += blockDim.x) {
        local_sum += matrix[row * m + i];
    }
    sdata[tid] = local_sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) {
        averages[row] = sdata[0] / m;
    }
}

extern "C" void gpu_propagate(float* d_in, float* d_out, int n, int m,
                               int block_x, int block_y, hipStream_t stream) {
    // Use texture memory
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = d_in;
    resDesc.res.pitch2D.desc = desc;
    resDesc.res.pitch2D.width = m;
    resDesc.res.pitch2D.height = n;
    resDesc.res.pitch2D.pitchInBytes = m * sizeof(float);

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipTextureObject_t tex = 0;
    CHECK_CUDA(hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr));

    dim3 threads(block_x, block_y);
    dim3 blocks((m + block_x - 1) / block_x, (n + block_y - 1) / block_y);
    propagate_kernel<<<blocks, threads, 0, stream>>>(tex, d_out, n, m);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDestroyTextureObject(tex));
}

extern "C" void gpu_calculate_averages(float* d_matrix, float* d_avg, int n, int m,
                                        int block_size, hipStream_t stream) {
    dim3 blocks(n);
    dim3 threads(block_size);
    size_t shared = block_size * sizeof(float);
    average_kernel<<<blocks, threads, shared, stream>>>(d_matrix, d_avg, n, m);
    CHECK_CUDA(hipGetLastError());
}

extern "C" void gpu_alloc_memory(float** d_in, float** d_out, int n, int m) {
    size_t bytes = n * m * sizeof(float);
    CHECK_CUDA(hipMalloc(d_in, bytes));
    CHECK_CUDA(hipMalloc(d_out, bytes));
}

extern "C" void gpu_free_memory(float* d_in, float* d_out) {
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
}

extern "C" void gpu_alloc_averages(float** d_avg, int n) {
    CHECK_CUDA(hipMalloc(d_avg, n * sizeof(float)));
}

extern "C" void gpu_free_averages(float* d_avg) {
    if (d_avg) hipFree(d_avg);
}

extern "C" void copy_to_device(float* d_in, const float* h_in, int n, int m) {
    CHECK_CUDA(hipMemcpy(d_in, h_in, n * m * sizeof(float), hipMemcpyHostToDevice));
}

extern "C" void copy_from_device(float* h_out, const float* d_out, int n, int m) {
    CHECK_CUDA(hipMemcpy(h_out, d_out, n * m * sizeof(float), hipMemcpyDeviceToHost));
}

extern "C" void copy_averages_from_device(float* h_avg, const float* d_avg, int n) {
    CHECK_CUDA(hipMemcpy(h_avg, d_avg, n * sizeof(float), hipMemcpyDeviceToHost));
}

extern "C" void validate_block_size(int block_x, int block_y) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    if (block_x <= 0 || block_y <= 0) {
        fprintf(stderr, "[ERROR] Block size must be positive: block_x = %d, block_y = %d\n",
                block_x, block_y);
        exit(EXIT_FAILURE);
    }

    if (block_x * block_y > prop.maxThreadsPerBlock) {
        fprintf(stderr, "[ERROR] Block size %d×%d exceeds max threads per block (%d)\n",
                block_x, block_y, prop.maxThreadsPerBlock);
        exit(EXIT_FAILURE);
    }
}

extern "C" void validate_results(const float* cpu_matrix, const float* gpu_matrix,
                                  const float* cpu_avg, const float* gpu_avg,
                                  int n, int m, bool has_avg) {
    float max_diff = 0.0f;
    int mismatch = 0;

    for (int i = 0; i < n * m; ++i) {
        float diff = fabs(cpu_matrix[i] - gpu_matrix[i]);
        if (diff > 1e-4f) mismatch++;
        max_diff = std::max(max_diff, diff);
    }
    printf("[Validation] Matrix max diff = %.6e, mismatches > 1e-4 = %d\n", max_diff, mismatch);

    if (has_avg) {
        float max_avg_diff = 0.0f;
        int avg_mismatch = 0;
        for (int i = 0; i < n; ++i) {
            float diff = fabs(cpu_avg[i] - gpu_avg[i]);
            if (diff > 1e-4f) avg_mismatch++;
            max_avg_diff = std::max(max_avg_diff, diff);
        }
        printf("[Validation] Averages max diff = %.6e, mismatches > 1e-4 = %d\n",
               max_avg_diff, avg_mismatch);
    }
}
