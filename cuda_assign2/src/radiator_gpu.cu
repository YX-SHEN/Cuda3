#include "hip/hip_runtime.h"
#include "radiator_gpu.h"
#include "cuda_helper.h"
#include <cmath>
#include <cstdio>
#include <algorithm>

// ---------------- Propagation Kernel with Shared Memory and Loop Unrolling ----------------
__global__ void propagate_kernel(const float* in, float* out, int n, int m) {
    extern __shared__ float tile[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;

    int local_row = ty;
    int local_col = tx + 2;

    // Tile width = blockDim.x + 4
    int shared_width = blockDim.x + 4;

    if (row < n && col < m)
        tile[local_row * shared_width + local_col] = in[row * m + col];

    if (tx < 2 && row < n) {
        int halo_col = (col - 2 + m) % m;
        tile[local_row * shared_width + tx] = in[row * m + halo_col];
    }

    if (tx >= blockDim.x - 2 && row < n) {
        int halo_col = (col + (tx - (blockDim.x - 2)) + 1) % m;
        tile[local_row * shared_width + local_col + (tx - (blockDim.x - 2)) + 1] =
            in[row * m + halo_col];
    }

    __syncthreads();

    if (row >= n || col >= m) return;

    if (col == 0) {
        out[row * m + col] = in[row * m + col];
        return;
    }

    float sum = 0.0f;
    #pragma unroll
    for (int offset = -2; offset <= 2; ++offset) {
        float coeff;
        switch (offset) {
            case -2: coeff = 1.60f; break;
            case -1: coeff = 1.55f; break;
            case  0: coeff = 1.00f; break;
            case  1: coeff = 0.60f; break;
            case  2: coeff = 0.25f; break;
        }
        sum += coeff * tile[local_row * shared_width + (local_col + offset)];
    }

    out[row * m + col] = sum / 5.0f;
}

// ---------------- Optimized Row Average Kernel ----------------
__global__ void average_kernel(const float* matrix, float* averages, int n, int m) {
    extern __shared__ float sdata[];
    int row = blockIdx.x;
    int tid = threadIdx.x;

    float local_sum = 0.0f;
    for (int i = tid; i < m; i += blockDim.x) {
        local_sum += matrix[row * m + i];
    }
    sdata[tid] = local_sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        averages[row] = sdata[0] / m;
    }
}

// ---------------- GPU API ----------------
extern "C"
void gpu_propagate(float* d_in, float* d_out, int n, int m,
                   int block_x, int block_y, hipStream_t stream) {
    dim3 threads(block_x, block_y);
    dim3 blocks((m + block_x - 1) / block_x, (n + block_y - 1) / block_y);
    size_t shared_mem = (block_x + 4) * block_y * sizeof(float);
    propagate_kernel<<<blocks, threads, shared_mem, stream>>>(d_in, d_out, n, m);
    CHECK_CUDA(hipGetLastError());
}

extern "C"
void gpu_calculate_averages(float* d_matrix, float* d_avg, int n, int m,
                            int block_size, hipStream_t stream) {
    dim3 blocks(n);
    dim3 threads(block_size);
    size_t shared = block_size * sizeof(float);
    average_kernel<<<blocks, threads, shared, stream>>>(d_matrix, d_avg, n, m);
    CHECK_CUDA(hipGetLastError());
}

// ---------------- Memory Management ----------------
extern "C"
void gpu_alloc_memory(float** d_in, float** d_out, int n, int m) {
    size_t bytes = n * m * sizeof(float);
    CHECK_CUDA(hipMalloc(d_in, bytes));
    CHECK_CUDA(hipMalloc(d_out, bytes));
}

extern "C"
void gpu_free_memory(float* d_in, float* d_out) {
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
}

extern "C"
void gpu_alloc_averages(float** d_avg, int n) {
    CHECK_CUDA(hipMalloc(d_avg, n * sizeof(float)));
}

extern "C"
void gpu_free_averages(float* d_avg) {
    if (d_avg) hipFree(d_avg);
}

extern "C"
void validate_block_size(int block_x, int block_y) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    if (block_x <= 0 || block_y <= 0) {
        fprintf(stderr, "[ERROR] Block size must be positive: block_x = %d, block_y = %d\n",
                block_x, block_y);
        exit(EXIT_FAILURE);
    }

    if (block_x * block_y > prop.maxThreadsPerBlock) {
        fprintf(stderr, "[ERROR] Block size %d×%d exceeds max threads per block (%d)\n",
                block_x, block_y, prop.maxThreadsPerBlock);
        exit(EXIT_FAILURE);
    }
}

// ---------------- Memory Transfers ----------------
extern "C"
void copy_to_device(float* d_in, const float* h_in, int n, int m) {
    CHECK_CUDA(hipMemcpy(d_in, h_in, n * m * sizeof(float), hipMemcpyHostToDevice));
}

extern "C"
void copy_from_device(float* h_out, const float* d_out, int n, int m) {
    CHECK_CUDA(hipMemcpy(h_out, d_out, n * m * sizeof(float), hipMemcpyDeviceToHost));
}

extern "C"
void copy_averages_from_device(float* h_avg, const float* d_avg, int n) {
    CHECK_CUDA(hipMemcpy(h_avg, d_avg, n * sizeof(float), hipMemcpyDeviceToHost));
}

// ---------------- Validation ----------------
extern "C"
void validate_results(const float* cpu_matrix, const float* gpu_matrix,
                      const float* cpu_avg, const float* gpu_avg,
                      int n, int m, bool has_avg) {
    float max_diff = 0.0f;
    int mismatch = 0;

    for (int i = 0; i < n * m; ++i) {
        float diff = fabs(cpu_matrix[i] - gpu_matrix[i]);
        if (diff > 1e-4f) mismatch++;
        max_diff = std::max(max_diff, diff);
    }
    printf("[Validation] Matrix max diff = %.6e, mismatches > 1e-4 = %d\n", max_diff, mismatch);

    if (has_avg) {
        float max_avg_diff = 0.0f;
        int avg_mismatch = 0;
        for (int i = 0; i < n; ++i) {
            float diff = fabs(cpu_avg[i] - gpu_avg[i]);
            if (diff > 1e-4f) avg_mismatch++;
            max_avg_diff = std::max(max_avg_diff, diff);
        }
        printf("[Validation] Averages max diff = %.6e, mismatches > 1e-4 = %d\n",
               max_avg_diff, avg_mismatch);
    }
}
